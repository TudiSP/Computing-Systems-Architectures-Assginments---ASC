#include "hip/hip_runtime.h"
#include "helper.h"
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

// aprox 1 mil
#define MSIZE (1 << 16)

__global__ void pop_in_range(float *lat, float *lon, int *pop, int *accpop,
                             int kmrange, int n) {
  unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  for (int j = 0; j < n; j++) {
    float phi1 = (90.f - lat[i]) * DEGREE_TO_RADIANS;
    float phi2 = (90.f - lat[j]) * DEGREE_TO_RADIANS;

    float theta1 = lon[i] * DEGREE_TO_RADIANS;
    float theta2 = lon[j] * DEGREE_TO_RADIANS;

    float cs =
        sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
    if (cs > 1) {
      cs = 1;
    } else if (cs < -1) {
      cs = -1;
    }
    if ((6371.f * acos(cs)) <= kmrange) {
      accpop[i] += pop[j];
    }
  }
}

int main(int argc, char **argv) {
  DIE(argc == 1, "./accpop <kmrange1> <file1in> <file1out> ...");
  DIE((argc - 1) % 3 != 0, "./accpop <kmrange1> <file1in> <file1out> ...");

  for (int argcID = 1; argcID < argc; argcID += 3) {
    float kmRange = atof(argv[argcID]);
    string geon;

    float *lat = (float *)malloc(MSIZE * sizeof(float));
    float *lon = (float *)malloc(MSIZE * sizeof(float));
    int *pop = (int *)malloc(MSIZE * sizeof(int));

    ifstream ifs(argv[argcID + 1]);
    ofstream ofs(argv[argcID + 2]);

    int i = 0;
    while (ifs >> geon >> lat[i] >> lon[i] >> pop[i]) {
      i++;
    }
    int nrOfElements = i;
    int *accpop = (int *)malloc(nrOfElements * sizeof(int));

    float *device_lat = 0;
    float *device_lon = 0;
    int *device_pop = 0, *device_accpop = 0;

    // Allocating the device's array; notice that we use a special
    // function named hipMalloc that takes the reference of the
    // pointer declared above and the number of bytes.
    hipMalloc((void **)&device_lat, nrOfElements * sizeof(float));
    hipMalloc((void **)&device_lon, nrOfElements * sizeof(float));
    hipMalloc((void **)&device_pop, nrOfElements * sizeof(int));
    hipMalloc((void **)&device_accpop, nrOfElements * sizeof(int));

    hipMemcpy(device_lat, lat, nrOfElements * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(device_lon, lon, nrOfElements * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(device_pop, pop, nrOfElements * sizeof(float),
               hipMemcpyHostToDevice);

    const size_t block_size = 256;
    size_t blocks_no = nrOfElements / block_size;

    if (nrOfElements % block_size)
      ++blocks_no;

    // Launch the kernel
    pop_in_range<<<blocks_no, block_size>>>(device_lat, device_lon, device_pop,
                                            device_accpop, kmRange,
                                            nrOfElements);

    // Copy the result back to the host memory space
    hipMemcpy(accpop, device_accpop, nrOfElements * sizeof(int),
               hipMemcpyDeviceToHost);

    for (i = 0; i < nrOfElements; i++) {
      ofs << accpop[i] << endl;
    }

    // Deallocate memory
    free(lat);
    free(lon);
    free(pop);
    free(accpop);
    hipFree(device_lat);
    hipFree(device_lon);
    hipFree(device_pop);
    hipFree(device_accpop);

    ifs.close();
    ofs.close();
  }
}
